#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "device_prop.cuh"
#include "utils.cuh"
#include "scatter_nd_of_shape_impl.cuh"
#include "cuda_type.h"

namespace contrib {

#define _ENFORCE(cond, msg) \
  if (!(cond)) ORTX_CXX_API_THROW(msg, ORT_RUNTIME_EXCEPTION);

#ifndef HIP_LONG
#define HIP_LONG int32_t
#endif

#ifndef CUDA_LONG
#define CUDA_LONG int32_t
#endif

template <typename T>
__device__ __forceinline__ void _add_inplace(T& x, const T a) { x += a; }

template <>
__device__ __forceinline__ void _add_inplace(half& x, const half a) {
#if __CUDA_ARCH__ < 700
  x = __float2half(__half2float(x) + __half2float(a));
#else
  x += a;
#endif
}

template <typename T>
__global__ void
addition_inplace_kernel(T* __restrict__ output_data, const int64_t* __restrict__ indices_data,
                        const T* __restrict__ updates_data, const CUDA_LONG indice_size,
                        const CUDA_LONG nrows, const CUDA_LONG stride) {
  HIP_LONG id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= stride)
    return;

  for (size_t i = 0; i < nrows; ++i) {
    output_data[i * stride + id] = 0;
  }

  int64_t index;
  for (size_t i = 0; i < indice_size; ++i) {
    index = (indices_data[i] + nrows) % nrows;
    _add_inplace(output_data[index * stride + id], updates_data[i * stride + id]);
  }
}

template <typename T>
__global__ void masked_addition_inplace_kernel(T *__restrict__ output_data,
                                               const int64_t *__restrict__ indices_data,
                                               const T *__restrict__ updates_data,
                                               const CUDA_LONG indice_size,
                                               const CUDA_LONG nrows, const CUDA_LONG stride,
                                               const int64_t masked_value) {
  auto id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= stride)
    return;

  for (size_t i = 0; i < nrows; ++i) {
    output_data[i * stride + id] = 0;
  }

  for (size_t i = 0; i < indice_size; ++i) {
    if (indices_data[i] == masked_value)
      continue;
    _add_inplace(output_data[indices_data[i] * stride + id], updates_data[i * stride + id]);
  }
}

template <typename T, int NTHREAD>
__global__ void masked_addition_inplace_kernelN(T *__restrict__ output_data,
                                                const int64_t *__restrict__ indices_data,
                                                const T *__restrict__ updates_data,
                                                const CUDA_LONG indice_size,
                                                const CUDA_LONG nrows, const CUDA_LONG stride,
                                                const int64_t masked_value) {
  __shared__ int64_t shared_indices[NTHREAD];

  CUDA_LONG tid = threadIdx.x;
  CUDA_LONG id = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t i = 0; i < nrows; ++i) {
    output_data[i * stride + id] = 0;
  }

  int begin = 0;
  int end = std::min(begin + NTHREAD, indice_size);
  while (begin < end && (end == begin + NTHREAD)) {
    shared_indices[tid] = indices_data[tid + begin];
    __syncthreads();

    for (size_t i = begin; i < end; ++i) {
      if (shared_indices[tid] == masked_value)
        continue;
      _add_inplace(output_data[shared_indices[tid] * stride + id],
                   updates_data[i * stride + id]);
    }

    begin = end;
    end = std::min(begin + NTHREAD, indice_size);
  }

  for (size_t i = begin; i < indice_size; ++i) {
    if (indices_data[i] == masked_value)
      continue;
    _add_inplace(output_data[indices_data[i] * stride + id], updates_data[i * stride + id]);
  }
}

template <class NTYPE>
NTYPE flattened_dimension(const std::vector<NTYPE>& values, size_t first = 0) {
  NTYPE r = 1;
  for (auto it = values.begin() + first; it != values.end(); ++it)
    r *= *it;
  return r;
}

template <typename T>
hipError_t ScatterNDOfShapeKernel(hipStream_t stream,
                                   const std::vector<int64_t>& output_shape,
                                   const std::vector<int64_t>& indices_shape,
                                   const int64_t* indices_data,
                                   const T* updates_data,
                                   T* output_data,
                                   ScatterReduction reduction) {
  if (reduction != ScatterReduction::Add)
    ORTX_CXX_API_THROW("Only reduction 'add' is implemented.", ORT_RUNTIME_EXCEPTION);
  size_t indice_size = static_cast<size_t>(flattened_dimension(indices_shape));
  size_t output_size = static_cast<size_t>(flattened_dimension(output_shape));
  size_t rank = output_shape.size() - indices_shape.size();
  size_t stride =  static_cast<size_t>(flattened_dimension(output_shape, output_shape.size() - 1 - rank));
  size_t nrows = output_size / stride;

  int threads_per_block = 256;
  int blocks_per_grid = (stride + threads_per_block - 1) / threads_per_block;

  dim3 threads(threads_per_block);
  dim3 blocks(blocks_per_grid);
  using TT = typename CudaT<T>::MappedType;
  addition_inplace_kernel<TT><<<blocks, threads, 0, stream>>>(reinterpret_cast<TT*>(output_data), indices_data,
                                                              reinterpret_cast<const TT*>(updates_data),
                                                              indice_size, nrows, stride);
  return hipGetLastError();
}

template <typename T>
hipError_t MaskedScatterNDOfShapeKernel(hipStream_t stream, const std::vector<int64_t> &input_shape,
                                         const std::vector<int64_t> &indices_shape,
                                         const int64_t *indices_data, const T *updates_data,
                                         T *output_data,
                                         ScatterReduction reduction, int64_t masked_value) {
  if (reduction != ScatterReduction::Add)
    ORTX_CXX_API_THROW("Only reduction 'add' is implemented.", ORT_RUNTIME_EXCEPTION);
  size_t indice_size = static_cast<size_t>(flattened_dimension(indices_shape));
  size_t input_size = static_cast<size_t>(flattened_dimension(input_shape));
  size_t stride = input_shape[input_shape.size() - 1];
  size_t nrows = input_size / stride;

  std::vector<size_t> next_batch(indice_size);
  std::vector<uint8_t> processed(input_shape[0], 0);
  std::vector<uint8_t> processed_once(input_shape[0], 0);

  int threads_per_block = 256;
  bool split = stride / threads_per_block <= 32;

  int blocks_per_grid = (stride + threads_per_block - 1) / threads_per_block;
  dim3 threads(threads_per_block);
  dim3 blocks(blocks_per_grid);

  using TT = typename CudaT<T>::MappedType;

  if (split && stride >= 256 && threads_per_block == 256) {
    masked_addition_inplace_kernelN<TT, 256><<<blocks, threads, 0, stream>>>(
        reinterpret_cast<TT*>(output_data), indices_data,
        reinterpret_cast<const TT*>(updates_data),
        indice_size, nrows, stride, masked_value);
  } else {
    masked_addition_inplace_kernel<TT><<<blocks, threads, 0, stream>>>(
        reinterpret_cast<TT*>(output_data), indices_data,
        reinterpret_cast<const TT*>(updates_data),
        indice_size, nrows, stride, masked_value);
  }
  return hipGetLastError();
}

template <>
hipError_t LaunchScatterNDOfShapeKernel<float>(hipStream_t stream,
                                                const std::vector<int64_t>& output_shape,
                                                const std::vector<int64_t>& indices_shape,
                                                const int64_t* indices,
                                                const float* updates,
                                                float* output,
                                                ScatterReduction reduction) {
  return ScatterNDOfShapeKernel(stream,
                                output_shape,
                                indices_shape,
                                indices,
                                updates,
                                output,
                                reduction);
}

template <>
hipError_t LaunchScatterNDOfShapeKernel<ortc::MFloat16>(hipStream_t stream,
                                                         const std::vector<int64_t>& output_shape,
                                                         const std::vector<int64_t>& indices_shape,
                                                         const int64_t* indices,
                                                         const ortc::MFloat16* updates,
                                                         ortc::MFloat16* output,
                                                         ScatterReduction reduction) {
  return ScatterNDOfShapeKernel(stream,
                                output_shape,
                                indices_shape,
                                indices,
                                updates,
                                output,
                                reduction);
}

template <>
hipError_t LaunchMaskedScatterNDOfShapeKernel<float>(hipStream_t stream,
                                                      const std::vector<int64_t>& output_shape,
                                                      const std::vector<int64_t>& indices_shape,
                                                      const int64_t* indices,
                                                      const float* updates,
                                                      float* output,
                                                      ScatterReduction reduction,
                                                      int64_t masked_value) {
  return MaskedScatterNDOfShapeKernel(stream,
                                      output_shape,
                                      indices_shape,
                                      indices,
                                      updates,
                                      output,
                                      reduction,
                                      masked_value);
}

template <>
hipError_t LaunchMaskedScatterNDOfShapeKernel<ortc::MFloat16>(hipStream_t stream,
                                                               const std::vector<int64_t>& output_shape,
                                                               const std::vector<int64_t>& indices_shape,
                                                               const int64_t* indices,
                                                               const ortc::MFloat16* updates,
                                                               ortc::MFloat16* output,
                                                               ScatterReduction reduction,
                                                               int64_t masked_value) {
  return MaskedScatterNDOfShapeKernel(stream,
                                      output_shape,
                                      indices_shape,
                                      indices,
                                      updates,
                                      output,
                                      reduction,
                                      masked_value);
}

}  // namespace contrib

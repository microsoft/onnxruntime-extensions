#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "device_prop.cuh"
#include "utils.cuh"
#include "mul_sigmoid_impl.cuh"
#include "cuda_type.h"

#ifndef CUDA_LONG
#define CUDA_LONG int32_t
#endif

using namespace Ort::Custom;

template <typename T> __device__ __inline__ T _exp_typed(const T x);

template <> __device__ __inline__ float _exp_typed(const float x) { return expf(x); }

#if __CUDA_ARCH__ < 700
template <> __device__ __inline__ half _exp_typed(const half x) {
  return __float2half(expf(__half2float(x)));
}
#else
template <> __device__ __inline__ half _exp_typed(const half x) { return hexp(x); }
#endif

template <typename T> __device__ __inline__ T sigmoid(const T a) {
  return a > T(0) ? (T)1 / ((T)1. + _exp_typed<T>(-a))
                  : (T)1 - (T)1 / ((T)1 + _exp_typed<T>(a));
}

#if __CUDA_ARCH__ < 700
template <> __device__ __inline__ half sigmoid(const half a) {
  return __float2half(sigmoid(__half2float(a)));
}
#endif

template <typename T> __device__ __inline__ T mul_sigmoid(const T a) { return a * sigmoid(a); }

#if __CUDA_ARCH__ < 700
template <> __device__ __inline__ half mul_sigmoid(const half a) {
  float x = __half2float(a);
  return __float2half(x * sigmoid(x));
}
#endif

template <typename T>
__global__ void MulSigmoidKernel(T *output_data, const T *input_data, CUDA_LONG N) {
  CUDA_LONG id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= N)
    return;
  output_data[id] = mul_sigmoid(input_data[id]);
}

template <typename T>
hipError_t _LaunchMulSigmoidKernel(hipStream_t stream, int input_length, const T* input, T* output) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  using TT = typename contrib::CudaT<T>::MappedType;
  MulSigmoidKernel<TT><<<gridSize, blockSize, 0, stream>>>(reinterpret_cast<TT*>(output), reinterpret_cast<const TT*>(input), input_length);
  return hipGetLastError();
}

template <>
hipError_t LaunchMulSigmoidKernel<float>(hipStream_t stream, int input_length, const float* input, float* output) {
  return _LaunchMulSigmoidKernel(stream, input_length, input, output);
}

template <>
hipError_t LaunchMulSigmoidKernel<ortc::MFloat16>(hipStream_t stream, int input_length, const ortc::MFloat16* input, ortc::MFloat16* output) {
  return _LaunchMulSigmoidKernel(stream, input_length, input, output);
}

#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "device_prop.cuh"
#include "utils.cuh"
#include "Rotary_impl.cuh"
#include "cuda_type.h"

using namespace Ort::Custom;

template <typename T> __device__ __inline__ T _neg(const T x) { return -x; }

#if __CUDA_ARCH__ < 700
template <> __device__ __inline__ half _neg(const half x) {
  return __float2half(-__half2float(x));
}
#endif

template <typename T, RotarySide side>
__global__ void RotaryKernel(T *output_data, const T *input_data, CUDA_LONG half_N, CUDA_LONG half_stride) {
  CUDA_LONG id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= half_N)
    return;
  CUDA_LONG last = id % half_stride;
  id = (id - last) * 2 + last;
  if (side == RotarySide::RIGHT) {
    output_data[id + half_stride] = input_data[id];
    output_data[id] = _neg(input_data[id + half_stride]);
  } else {
    output_data[id + half_stride] = _neg(input_data[id]);
    output_data[id] = input_data[id + half_stride];
  }
}

template <typename T>
hipError_t _LaunchRotaryKernel(hipStream_t stream, int input_length, int last_dim,
                                const T* input, const int64_t* split_data, T* output, RotarySide side) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  if (input_length == 0)
      return;
  using TT = typename contrib::CudaT<T>::MappedType;

  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  CUDA_LONG stride = static_cast<CUDA_LONG>(last_dim);

  const int num_threads_per_block = GridDim::maxThreadsPerBlock;
  const int num_elements_per_thread =
      (N / 2 + num_threads_per_block - 1) / num_threads_per_block;

  switch (side) {
  case RotarySide::LEFT:
    RotaryKernel<T, RotarySide::LEFT>
        <<<num_elements_per_thread, num_threads_per_block, 0, stream>>>(output_data, input_data,
                                                                        N / 2, stride / 2);
    break;
  case RotarySide::RIGHT:
    RotaryKernel<T, RotarySide::RIGHT>
        <<<num_elements_per_thread, num_threads_per_block, 0, stream>>>(output_data, input_data,
                                                                        N / 2, stride / 2);
    break;
  }

  RotaryKernel<TT><<<gridSize, blockSize, 0, stream>>>(reinterpret_cast<TT*>(output), reinterpret_cast<const TT*>(input), input_length);
  return hipGetLastError();
}

template <>
hipError_t LaunchRotaryKernel<float>(hipStream_t stream, int input_length, int last_dim,
                                      const float* input, const int64_t* split_data, float* output, RotarySide side) {
  return _LaunchRotaryKernel(stream, input_length, last_dim, input, split_data, output, side);
}

template <>
hipError_t LaunchRotaryKernel<ortc::MFloat16>(hipStream_t stream, int input_length, int last_dim,
                                               const ortc::MFloat16* input, const int64_t* split_data,
                                               ortc::MFloat16* output, RotarySide side) {
  return _LaunchRotaryKernel(stream, input_length, last_dim, input, split_data, output, side);
}


#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#if OCOS_USE_FLASH_ATTENTION

#include "flash_fwd_launch_template.h"

namespace flash {

template void run_mha_fwd_splitkv_dispatch<cutlass::bfloat16_t, 32>(Flash_fwd_params& params, cudaStream_t stream);

}  // namespace flash
#endif

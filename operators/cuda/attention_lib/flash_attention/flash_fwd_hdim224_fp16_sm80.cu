
#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
#if OCOS_USE_FLASH_ATTENTION

#include "flash_fwd_launch_template.h"

namespace flash {

template <>
void run_mha_fwd_<cutlass::half_t, 224>(Flash_fwd_params& params, cudaStream_t stream) {
  run_mha_fwd_hdim224<cutlass::half_t>(params, stream);
}

}  // namespace flash
#endif


#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
#if OCOS_USE_FLASH_ATTENTION

#include "flash_fwd_launch_template.h"

namespace flash {

template <>
void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params& params, cudaStream_t stream) {
  run_mha_fwd_hdim128<cutlass::half_t>(params, stream);
}

}  // namespace flash
#endif

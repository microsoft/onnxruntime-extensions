#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "fast_gelu_impl.cuh"
#include <hip/hip_runtime.h>

template <typename T>
__device__ __inline T _Tanh(T a);

template <>
__device__ __inline__ float _Tanh(float a) { return tanhf(a); }

constexpr float A = 0.5f;

constexpr float B = 0.7978845608028654f;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125f;  // 0.044715 * sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, int input_length, int bias_length,
                               const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (T)(x + bias[idx % bias_length]);
    const T cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <>
hipError_t LaunchFastGeluKernel(hipStream_t stream, int input_length, int bias_length,
                                 const float* input, const float* bias, float* output, bool /*use_half2*/) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  FastGeluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length,
                                                                       input, bias, output);

  return hipGetLastError();
}
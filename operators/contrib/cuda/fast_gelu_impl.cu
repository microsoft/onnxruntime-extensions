#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "onnxruntime_f16.h"
#include "fast_gelu_impl.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

using namespace Ort::Custom;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 530) && ((__CUDACC_VER_MAJOR__ < 12) || ((__CUDACC_VER_MAJOR__ == 12) && (__CUDACC_VER_MINOR__ < 2)))
__device__ __forceinline__ half operator+(const half& lh, const half& rh) { return half((float)lh + (float)rh); }
__device__ __forceinline__ half operator-(const half& lh, const half& rh) { return half((float)lh - (float)rh); }
__device__ __forceinline__ half operator*(const half& lh, const half& rh) { return half((float)lh * (float)rh); }
__device__ __forceinline__ half operator/(const half& lh, const half& rh) { return half((float)lh / (float)rh); }

__device__ __forceinline__ half& operator+=(half& lh, const half& rh) {
  lh = half((float)lh + (float)rh);
  return lh;
}
__device__ __forceinline__ half& operator-=(half& lh, const half& rh) {
  lh = half((float)lh - (float)rh);
  return lh;
}
__device__ __forceinline__ half& operator*=(half& lh, const half& rh) {
  lh = half((float)lh * (float)rh);
  return lh;
}
__device__ __forceinline__ half& operator/=(half& lh, const half& rh) {
  lh = half((float)lh / (float)rh);
  return lh;
}

/* Note for increment and decrement we use the raw value 0x3C00 equating to half(1.0f), to avoid the extra conversion */
__device__ __forceinline__ __half& operator++(__half& h) {
  h = half((float)h + 1.0f);
  return h;
}
__device__ __forceinline__ __half& operator--(__half& h) {
  h = half((float)h - 1.0f);
  return h;
}
__device__ __forceinline__ __half operator++(__half& h, int) {
  half ret = h;
  h = half((float)h + 1);
  return ret;
}
__device__ __forceinline__ __half operator--(__half& h, int) {
  half ret = h;
  h = half((float)h - 1);
  return ret;
}

/* Unary plus and inverse operators */
__device__ __forceinline__ half operator+(const half& h) { return h; }
__device__ __forceinline__ half operator-(const half& h) { return half(-(float)h); }

/* Some basic comparison operations to make it look like a builtin */
__device__ __forceinline__ bool operator==(const half& lh, const half& rh) { return (float)lh == (float)rh; }
__device__ __forceinline__ bool operator!=(const half& lh, const half& rh) { return (float)lh != (float)rh; }
__device__ __forceinline__ bool operator>(const half& lh, const half& rh) { return (float)lh > (float)rh; }
__device__ __forceinline__ bool operator<(const half& lh, const half& rh) { return (float)lh < (float)rh; }
__device__ __forceinline__ bool operator>=(const half& lh, const half& rh) { return (float)lh >= (float)rh; }
__device__ __forceinline__ bool operator<=(const half& lh, const half& rh) { return (float)lh <= (float)rh; }

// support half2 arithmetic for cuda architecture < 5.3
__device__ __forceinline__ half2 operator+(const half2& lh, const half2& rh) {
  half2 r;
  r.x = lh.x + rh.x;
  r.y = lh.y + rh.y;
  return r;
}

__device__ __forceinline__ half2 operator-(const half2& lh, const half2& rh) {
  half2 r;
  r.x = lh.x - rh.x;
  r.y = lh.y - rh.y;
  return r;
}

__device__ __forceinline__ half2 operator*(const half2& lh, const half2& rh) {
  half2 r;
  r.x = lh.x * rh.x;
  r.y = lh.y * rh.y;
  return r;
}

__device__ __forceinline__ half2 operator/(const half2& lh, const half2& rh) {
  half2 r;
  r.x = lh.x / rh.x;
  r.y = lh.y / rh.y;
  return r;
}
#endif

/// Arithmetic for BFloat16

__device__ __forceinline__ BFloat16 operator+(const BFloat16& a, const BFloat16& b) {
  return static_cast<float>(a) + static_cast<float>(b);
}

__device__ __forceinline__ BFloat16 operator-(const BFloat16& a, const BFloat16& b) {
  return static_cast<float>(a) - static_cast<float>(b);
}

__device__ __forceinline__ BFloat16 operator*(const BFloat16& a, const BFloat16& b) {
  return static_cast<float>(a) * static_cast<float>(b);
}

__device__ __forceinline__ BFloat16 operator/(const BFloat16& a, const BFloat16& b) {
  return static_cast<float>(a) / static_cast<float>(b);
}

__device__ __forceinline__ BFloat16 operator-(const BFloat16& a) { return -static_cast<float>(a); }

__device__ __forceinline__ BFloat16& operator+=(BFloat16& a, const BFloat16& b) {
  a = a + b;
  return a;
}

__device__ __forceinline__ BFloat16& operator-=(BFloat16& a, const BFloat16& b) {
  a = a - b;
  return a;
}

__device__ __forceinline__ BFloat16& operator*=(BFloat16& a, const BFloat16& b) {
  a = a * b;
  return a;
}

__device__ __forceinline__ BFloat16& operator/=(BFloat16& a, const BFloat16& b) {
  a = a / b;
  return a;
}

/// Arithmetic with floats

__device__ __forceinline__ float operator+(BFloat16 a, float b) { return a + b; }
__device__ __forceinline__ float operator-(BFloat16 a, float b) { return a - b; }
__device__ __forceinline__ float operator*(BFloat16 a, float b) { return a * b; }
__device__ __forceinline__ float operator/(BFloat16 a, float b) { return a / b; }

__device__ __forceinline__ float operator+(float a, BFloat16 b) { return a + b; }
__device__ __forceinline__ float operator-(float a, BFloat16 b) { return a - b; }
__device__ __forceinline__ float operator*(float a, BFloat16 b) { return a * b; }
__device__ __forceinline__ float operator/(float a, BFloat16 b) { return a / b; }

__device__ __forceinline__ float& operator+=(float& a, const BFloat16& b) { return a += b; }
__device__ __forceinline__ float& operator-=(float& a, const BFloat16& b) { return a -= b; }
__device__ __forceinline__ float& operator*=(float& a, const BFloat16& b) { return a *= b; }
__device__ __forceinline__ float& operator/=(float& a, const BFloat16& b) { return a /= b; }

/// Arithmetic with doubles

__device__ __forceinline__ double operator+(BFloat16 a, double b) { return static_cast<double>(a) + b; }
__device__ __forceinline__ double operator-(BFloat16 a, double b) { return static_cast<double>(a) - b; }
__device__ __forceinline__ double operator*(BFloat16 a, double b) { return static_cast<double>(a) * b; }
__device__ __forceinline__ double operator/(BFloat16 a, double b) { return static_cast<double>(a) / b; }

__device__ __forceinline__ double operator+(double a, BFloat16 b) { return a + static_cast<double>(b); }
__device__ __forceinline__ double operator-(double a, BFloat16 b) { return a - static_cast<double>(b); }
__device__ __forceinline__ double operator*(double a, BFloat16 b) { return a * static_cast<double>(b); }
__device__ __forceinline__ double operator/(double a, BFloat16 b) { return a / static_cast<double>(b); }

// Overloading < and > operators

__device__ __forceinline__ bool operator==(BFloat16& lhs, BFloat16& rhs) { return float(lhs) == float(rhs); }
__device__ __forceinline__ bool operator!=(BFloat16& lhs, BFloat16& rhs) { return float(lhs) != float(rhs); }
__device__ __forceinline__ bool operator>(BFloat16& lhs, BFloat16& rhs) { return float(lhs) > float(rhs); }
__device__ __forceinline__ bool operator<(BFloat16& lhs, BFloat16& rhs) { return float(lhs) < float(rhs); }

template <typename T>
__device__ __inline T _Tanh(T a);

template <>
__device__ __inline__ float _Tanh(float a) { return tanhf(a); }

template <>
__device__ __inline__ half _Tanh(half a) { return half(tanhf((float)a)); }

template <>
__device__ __inline__ half2 _Tanh(half2 a) {
  float2 tmp = (__half22float2(a));
  tmp.x = tanhf(tmp.x);
  tmp.y = tanhf(tmp.y);
  return __float22half2_rn(tmp);
}

template <>
__device__ __inline__ BFloat16 _Tanh(BFloat16 a) { return tanhf(static_cast<float>(a)); }

constexpr float A = 0.5f;

constexpr float B = 0.7978845608028654f;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125f;  // 0.044715 * sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, int input_length, int bias_length,
                               const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (T)(x + bias[idx % bias_length]);
    const T cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel2(const half2 a, const half2 b, const half2 c, int input_length, int bias_length,
                                const half2* input, const half2* bias, half2* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < input_length) {
    const half2 x = input[idx];
    const half2 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const half2 cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <>
hipError_t LaunchFastGeluKernel(hipStream_t stream, int input_length, int bias_length,
                                 const float* input, const float* bias, float* output, bool /*use_half2*/) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  FastGeluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length,
                                                                       input, bias, output);

  return hipGetLastError();
}

template <>
hipError_t LaunchFastGeluKernel(hipStream_t stream, int input_length, int bias_length,
                                 const half* input, const half* bias, half* output, bool use_half2) {
  constexpr int blockSize = 256;
  if (use_half2 && 0 == (bias_length & 1) /*&& prop.major >= 7*/ ) { // todo - get device id from ort for device property
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const half2 A2 = __floats2half2_rn(A, A);
    const half2 B2 = __floats2half2_rn(B, B);
    const half2 C2 = __floats2half2_rn(C, C);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    FastGeluKernel2<blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n, bias_length / 2,
                                                                   input2, bias2, output2);
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    FastGeluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length,
                                                                        input, bias, output);
  }

  return hipGetLastError();
}

template <>
hipError_t LaunchFastGeluKernel(hipStream_t stream, int input_length, int bias_length,
                                 const BFloat16* input, const BFloat16* bias, BFloat16* output, bool /*use_half2*/) {
  constexpr int blockSize = 256;

  // remove nv_bfloat162 implementation for now to fix build issue
  // we can decide whether to add it back if there's perf concern
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  FastGeluKernel<BFloat16, blockSize>
      <<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);

  return hipGetLastError();
}